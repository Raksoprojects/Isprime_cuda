#include <hip/hip_runtime.h>
#include <math.h>
#include <chrono>
#include <iostream>

#define N 1024
typedef unsigned long long int int64_cu;

__global__ void is_prime(int64_cu *prime, int64_cu *root, bool *c){
    if(*prime<2) {*c = false; return;}
    if(*prime % 2 == 0) {*c = false; return;}

    for(int64_cu k = 3+(2*blockIdx.x); k <= *root; k = k + 2 * N){
        if(*prime % k == 0) {*c = false; return ;}
        
    }
    *c = true;
    return ;
}

bool is_prime(int64_cu p){

    if(p<2) return false;
    if(p % 2 == 0) return false;

    for(int64_cu k = 3; k <= sqrt(p); k = k + 2){
        if(p % k == 0) return false;
    }
    return true;
}

int main(){
    
    ///Kod na CPU
    int64_cu test = 0;
    std::cout<<"Type a number to check if it is prime: ";
    std::cin>>test;
    auto t_start = std::chrono::high_resolution_clock::now();
    if(is_prime(test)){
        std::cout<<"It is a prime!";
    }
    else{
        std::cout<<"It is not a prime!";
    }
    auto t_end = std::chrono::high_resolution_clock::now();

    double elapsed_time_ms = std::chrono::duration<double, std::milli>(t_end-t_start).count();
    std::cout<<"It took: "<< elapsed_time_ms << " ms";

    ///Kod na GPU
    int64_cu prime, root; 
    bool czyjest;
    int64_cu *d_prime, *d_root;
    bool *czy;
    int64_cu size = N * sizeof(int64_cu);
    int size2 = sizeof(bool);

    ///Allocate space for devices copies
    hipMalloc((void **)&d_prime, size);
    hipMalloc((void **)&d_root, size);
    hipMalloc((void **)&czy, size2);

    std::cout<<"Podaj liczbę do sprawdzenia: ";
    std::cin>>prime;
    root = sqrt(prime);

    t_start = std::chrono::high_resolution_clock::now();

    hipMemcpy(d_prime, &prime, size, hipMemcpyHostToDevice);
    hipMemcpy(d_root, &root, size, hipMemcpyHostToDevice);
    
    is_prime<<<N,1>>>(d_prime, d_root, czy);

    hipMemcpy(&czyjest, czy, size2, hipMemcpyDeviceToHost);

    hipFree(d_prime); hipFree(czy); hipFree(d_root);
    if(czyjest){
        std::cout<<"To jest liczba pierwsza!";
    }
    else std::cout<<"To nie jest liczba pierwsza!";

    t_end = std::chrono::high_resolution_clock::now();
    
    elapsed_time_ms = std::chrono::duration<double, std::milli>(t_end-t_start).count();
    std::cout<<"It took: "<<elapsed_time_ms<<" ms!";

    return 0;
}